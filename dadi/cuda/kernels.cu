
#include <hip/hip_runtime.h>
__global__ void inject_mutations_2D(double *phi, int L, double val01, double val10){
    phi[1] += val01;
    phi[L] += val10;
}

__global__ void inject_mutations_3D(double *phi, int L, double val001, double val010, double val100){
    phi[1] += val001;
    phi[L] += val010;
    phi[L*L] += val100;
}

__global__ void Vfunc(double *x, double nu, int L, double *output){
    int ii = blockIdx.x*blockDim.x + threadIdx.x;
    if(ii < L){
        output[ii] =  x[ii] * (1.-x[ii])/nu;
    }
}

__device__ double _Mfunc2D(double x, double y, double m, double gamma, double h){
    return m * (y-x) + gamma * 2*(h + (1.-2*h)*x) * x*(1.-x);
}

__global__ void Mfunc2D(double *x, double *y, double m, double gamma, double h, int L, int M, double *output){
    int ii = (blockIdx.x*blockDim.x + threadIdx.x) / M;
    int jj = (blockIdx.x*blockDim.x + threadIdx.x) % M;
    if(ii < L){
        output[ii*M + jj] = _Mfunc2D(x[ii], y[jj], m, gamma, h);
    }
}

__device__ double _Mfunc3D(double x, double y, double z, double mxy, double mxz,
        double gamma, double h){
    return mxy * (y-x) + mxz * (z-x) + gamma * 2*(h + (1.-2*h)*x) * x*(1.-x);
}

__global__ void Mfunc3D(double *x, double *y, double *z, double mxy, double mxz, double gamma, double h, int L, int M, int N, double *output){
    int ii = (blockIdx.x*blockDim.x + threadIdx.x) / (M*N);
    int jj = ((blockIdx.x*blockDim.x + threadIdx.x) / N) % M;
    int kk = (blockIdx.x*blockDim.x + threadIdx.x) % N;
    if(ii < L){
        output[ii*(M*N) + jj*N + kk] = _Mfunc3D(x[ii], y[jj], z[kk], mxy, mxz, gamma, h);
    }
}

// We need an additional simple kernel to zero out the necessary
// values of the c array, because the Interleaved tridiagonal
// solver alters the c array.
__global__ void cx0(double *cx, int L, int M){
    int jj = blockIdx.x*blockDim.x + threadIdx.x;
    if(jj < M){
        cx[(L-1)*M + jj] = 0;
    }
}

// This function works for 2D and 3D cases, because migration terms
// don't matter at the 0,0,0 and 1,1,1 corners of the regime.
__global__ void include_bc(double*dx, double nu1, double gamma, double h, int L, int M, double *b){
    double Mfirst, Mlast;
    // 0,0 entry
    Mfirst = _Mfunc2D(0, 0, 0, gamma, h);
    if(Mfirst <= 0){
        b[0] += (0.5/nu1 - Mfirst)*2./dx[0];
    }
    // -1,-1 entry
    Mlast = _Mfunc2D(1, 1, 0, gamma, h);
    if(Mlast >= 0){
        b[L*M-1] += -(-0.5/nu1 - Mlast)*2./dx[L-2];
    }
}

// Compared to the C code, we need to separate ab and bc
// calculations, to avoid a race condition due to multiple
// theads writing to the same elements of b.
__global__ void compute_ab_nobc(double *dx, double *dfactor, 
        double *MInt, double *V, double dt, int L, int M,
        double *a, double *b){
    int ii = (blockIdx.x*blockDim.x + threadIdx.x) / M;
    int jj = (blockIdx.x*blockDim.x + threadIdx.x) % M;
    double atemp;

    if(ii < L-1){
        atemp = MInt[ii*M + jj] * 0.5 + V[ii]/(2*dx[ii]);
        a[(ii+1)*M + jj] = -dfactor[ii+1]*atemp;
        b[ii*M + jj] += dfactor[ii]*atemp;
    }
}

__global__ void compute_bc_nobc(double *dx, double *dfactor, 
        double *MInt, double *V, double dt, int L, int M,
        double *b, double *c){
    int ii = (blockIdx.x*blockDim.x + threadIdx.x) / M;
    int jj = (blockIdx.x*blockDim.x + threadIdx.x) % M;
    double ctemp;

    if(ii < L-1){
        ctemp = -MInt[ii*M + jj] * 0.5 + V[ii+1]/(2*dx[ii]);
        b[(ii+1)*M + jj] += dfactor[ii+1]*ctemp;
        c[ii*M + jj] = -dfactor[ii]*ctemp;
    }
}